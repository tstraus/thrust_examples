#include "hip/hip_runtime.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
//#include <thrust/generate.h>
//#include <thrust/reduce.h>
//#include <thrust/functional.h>
//#include <thrust/random.h>
//
//int my_rand (void)
//{
//    static thrust::default_random_engine rng;
//    static thrust::uniform_int_distribution<int> dist(0, 9999);
//
//    return dist(rng);
//}
//
//int main(void)
//{
//    for (int i = 0; i < 100; i++)
//    {
//        // generate random data on the host
//        thrust::host_vector<int> h_vec(1000000);
//        thrust::generate(h_vec.begin(), h_vec.end(), my_rand);
//
//        // transfer to device and compute sum
//        thrust::device_vector<int> d_vec = h_vec;
//
//        // initial value of the reduction
//        int init = 0; 
//
//        // binary operation used to reduce values
//        thrust::plus<int> binary_op;
//
//        // compute sum on the device
//        int sum = thrust::reduce(d_vec.begin(), d_vec.end(), init, binary_op);
//
//        // print the sum
//        std::cout << "sum is " << sum << std::endl;
//    }
//
//    return 0;
//}

#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include <iostream>
#include <iomanip>
#include <cmath>

// we could vary M & N to find the perf sweet spot

__host__ __device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);

    return a;
}

struct estimate_pi : public thrust::unary_function<unsigned int,float>
{
    __host__ __device__
    float operator()(unsigned int thread_id)
    {
        float sum = 0;
        unsigned int N = 1000000; // samples per thread

        unsigned int seed = hash(thread_id);

        // seed a random number generator
        thrust::default_random_engine rng(seed);

        // create a mapping from random numbers to [0,1)
        thrust::uniform_real_distribution<float> u01(0,1);

        // take N samples in a quarter circle
        for(unsigned int i = 0; i < N; ++i)
        {
            // draw a sample from the unit square
            float x = u01(rng);
            float y = u01(rng);

            // measure distance from the origin
            float dist = sqrtf(x*x + y*y);

            // add 1.0f if (u0,u1) is inside the quarter circle
            if(dist <= 1.0f)
            sum += 1.0f;
        }

        // multiply by 4 to get the area of the whole circle
        sum *= 4.0f;

        // divide by N
        return sum / N;
    }
};

int main(void)
{
    // use 30K independent seeds
    int M = 30000;

    float estimate = thrust::transform_reduce(thrust::counting_iterator<int>(0),
    thrust::counting_iterator<int>(M),
    estimate_pi(),
    0.0f,
    thrust::plus<float>());

    estimate /= M;

    std::cout << std::setprecision(16);
    std::cout << "pi is approximately " << estimate << std::endl;

    return 0;
}

